#include "common.hpp"

#include "hipsolver.h"

#ifdef USE_COMPLEX
#ifdef USE_FLOAT
#define hipsolverDnXpotrf_bufferSize hipsolverDnCpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnCpotrf
#define cusolverDnX hipComplex
#else // USE_DOUBLE
#define hipsolverDnXpotrf_bufferSize hipsolverDnZpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnZpotrf
#define cusolverDnX hipDoubleComplex
#endif // USE_FLOAT
#else // USE_REAL
#ifdef USE_FLOAT
#define hipsolverDnXpotrf_bufferSize hipsolverDnSpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnSpotrf
#define cusolverDnX float
#else // USE_DOUBLE
#define hipsolverDnXpotrf_bufferSize hipsolverDnDpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnDpotrf
#define cusolverDnX double
#endif // USE_FLOAT
#endif // USE_COMPLEX

static int Nmin = 0, Nmax = 0, Nstep = 0, _samples = 0, device_ = 0, _devices = 0;
static int lda = 0, Lwork = 0;

static hipsolverStatus_t status;
static hipsolverHandle_t handle;

static cusolverDnX *Agpu = (cusolverDnX*)NULL, *Workspace = (cusolverDnX*)NULL, *Acpu = (cusolverDnX*)NULL, *wrk = (cusolverDnX*)NULL;

static void device_count()
{
  const hipError_t error = hipGetDeviceCount(&_devices); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorNoDevice:
    (void)fprintf(stderr, "[%s@%s:%d] NoDevice\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorInsufficientDriver:
    (void)fprintf(stderr, "[%s@%s:%d] InsufficientDriver\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
}

static void set_device()
{
  int device = 0;
  (void)hipGetDevice(&device);
  if (device != device_) {
    const hipError_t error = hipSetDevice(device_); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevice:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevice\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorContextAlreadyInUse:
      (void)fprintf(stderr, "[%s@%s:%d] DeviceAlreadyInUse\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
  }
}

static void create_handle()
{
  status = hipsolverDnCreate(&handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ALLOC_FAILED:
    (void)fprintf(stderr, "[%s@%s:%d] ALLOC_FAILED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ARCH_MISMATCH:
    (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

static void alloc_gpu_mtx()
{
  size_t pitch = 0;
  const hipError_t error = hipMallocPitch(&Agpu, &pitch, Nmax * sizeof(cusolverDnX), Nmax); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorOutOfMemory:
    (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  lda = int(pitch / sizeof(cusolverDnX));
#ifndef NDEBUG
  (void)fprintf(stdout, "lda = %d\n", lda);
#endif // !NDEBUG
}

static void find_lwork()
{
  int LworkL = -1, maxLworkL = 0;
  int LworkU = -1, maxLworkU = 0;

  for (int n = Nmin; n <= Nmax; n += Nstep) {
    status = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, Agpu, lda, &LworkL); const int lin1 = __LINE__;
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      // (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, status);
      exit(status);
    }
    if (LworkL > maxLworkL)
      maxLworkL = LworkL;
    status = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, Agpu, lda, &LworkU); const int lin2 = __LINE__;
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      // (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, status);
      exit(status);
    }
    if (LworkU > maxLworkU)
      maxLworkU = LworkU;
  }

  Lwork = ((maxLworkL >= maxLworkU) ? maxLworkL : maxLworkU);
#ifndef NDEBUG
  (void)fprintf(stdout, "Lwork = %d, LworkL = %d, LworkU = %d\n", Lwork, maxLworkL, maxLworkU);
#endif // !NDEBUG
}

static void alloc_gpu_wrk()
{
  if (Lwork > 0) {
    const hipError_t error = hipMalloc(&Workspace, Lwork * sizeof(cusolverDnX)); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
  }
}

static void alloc_cpu_mtx()
{
  const size_t size = size_t(lda) * Nmax * sizeof(cusolverDnX);
  if (size > 0) {
    const hipError_t error = hipHostMalloc(&Acpu, size); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }   (void)memset(Acpu, 0, size);
    wrk = (cusolverDnX*)calloc(3 * Nmax, sizeof(cusolverDnX)); const int lin2 = __LINE__;
    if (!wrk) {
      (void)fprintf(stderr, "[%s@%s:%d] ", __FUNCTION__, __FILE__, lin2);
      perror((const char*)NULL);
      exit(errno);
    }
  }
}

static double init_cpu_mtx()
{
  static const int idist = 1;
  int iseed[4] = { 0, 1, 2, 3 };
  const int k = Nmax - 1;
  int info = 0;

  const double go = omp_get_wtime();

  // diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, wrk);
  // A
  REAL_LAPACK(lagsy)(&Nmax, &k, wrk, Acpu, &lda, iseed, wrk + Nmax, &info); const int lin = __LINE__;
  if (info) {
    (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, info);
    exit(info);
  }

  return (omp_get_wtime() - go);
}

static double copy_mtx_cpu2gpu()
{
  const double go = omp_get_wtime();
  // TODO: hipMemcpy2D
  return (omp_get_wtime() - go);
}

static void free_cpu_mtx()
{
  if (Acpu) {
    const hipError_t error = hipHostFree(Acpu); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Acpu = (cusolverDnX*)NULL;
  }
}

static void free_gpu_wrk()
{
  if (Workspace) {
    const hipError_t error = hipFree(Workspace); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Workspace = (cusolverDnX*)NULL;
  }
}

static void free_gpu_mtx()
{
  const hipError_t error = hipFree(Agpu); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorNotInitialized:
    (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
    exit(error);    
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  Agpu = (cusolverDnX*)NULL;
}

static void destroy_handle()
{
  status = hipsolverDnDestroy(handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

int main(int argc, char* argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s Nmin Nmax Nstep #samples [device#]\n", *argv);
    return EXIT_FAILURE;
  }

  if ((Nmin = atoi(argv[1])) <= 0) {
    (void)fprintf(stderr, "Nmin == %d <= 0\n", Nmin);
    return EXIT_FAILURE;
  }

  if ((Nmax = atoi(argv[2])) <= 0) {
    (void)fprintf(stderr, "Nmax == %d <= 0\n", Nmax);
    return EXIT_FAILURE;
  }

  if (Nmax < Nmin) {
    (void)fprintf(stderr, "Nmax == %d < Nmin == %d\n", Nmax, Nmin);
    return EXIT_FAILURE;
  }

  if ((Nstep = atoi(argv[3])) <= 0) {
    (void)fprintf(stderr, "Nstep == %d <= 0\n", Nstep);
    return EXIT_FAILURE;
  }

  if ((_samples = atoi(argv[4])) <= 0) {
    (void)fprintf(stderr, "#samples == %d <= 0\n", _samples);
    return EXIT_FAILURE;
  }

  if ((argc == 6) && ((device_ = atoi(argv[5])) < 0)) {
    (void)fprintf(stderr, "device# == %d < 0\n", device_);
    return EXIT_FAILURE;
  }

#ifndef NDEBUG
  (void)fprintf(stdout, "[omp_get_wtick] %#.17E s\n", omp_get_wtick());
#endif // !NDEBUG

  device_count();
  if (device_ > _devices) {
    (void)fprintf(stderr, "device# == %d > #devices == %d\n", device_, _devices);
    return EXIT_FAILURE;
  }
  set_device();

  create_handle();
  alloc_gpu_mtx();
  find_lwork();
  alloc_gpu_wrk();
  alloc_cpu_mtx();
  const double init_time = init_cpu_mtx();
#ifndef NDEBUG
  (void)fprintf(stdout, "[init_cpu_mtx] %#.17E s\n", init_time);
#endif // !NDEBUG
  const double copy_time = copy_mtx_cpu2gpu();
#ifndef NDEBUG
  (void)fprintf(stdout, "[copy_mtx_cpu2gpu] %#.17E s\n", copy_time);
#endif // !NDEBUG
  free_cpu_mtx();
  free_gpu_wrk();
  free_gpu_mtx();
  destroy_handle();

  return EXIT_SUCCESS;
}
