#include "common.hpp"

#include "hipsolver.h"

#ifdef USE_COMPLEX
#ifdef USE_FLOAT
#define hipsolverDnXpotrf_bufferSize hipsolverDnCpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnCpotrf
#define cusolverDnX hipComplex
#else // USE_DOUBLE
#define hipsolverDnXpotrf_bufferSize hipsolverDnZpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnZpotrf
#define cusolverDnX hipDoubleComplex
#endif // USE_FLOAT
#else // USE_REAL
#ifdef USE_FLOAT
#define hipsolverDnXpotrf_bufferSize hipsolverDnSpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnSpotrf
#define cusolverDnX float
#else // USE_DOUBLE
#define hipsolverDnXpotrf_bufferSize hipsolverDnDpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnDpotrf
#define cusolverDnX double
#endif // USE_FLOAT
#endif // USE_COMPLEX

static int Nmin = 0, Nmax = 0, Nstep = 0, _samples = 0, device_ = 0, _devices = 0;
static int lda = 0, Lwork = 0;

static hipsolverStatus_t status;
static hipsolverHandle_t handle;

static cusolverDnX *Agpu = (cusolverDnX*)NULL, *Workspace = (cusolverDnX*)NULL, *Acpu = (cusolverDnX*)NULL, *wrk = (cusolverDnX*)NULL;

static void device_count()
{
  const hipError_t error = hipGetDeviceCount(&_devices); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorNoDevice:
    (void)fprintf(stderr, "[%s@%s:%d] NoDevice\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorInsufficientDriver:
    (void)fprintf(stderr, "[%s@%s:%d] InsufficientDriver\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
}

static void set_device()
{
  int device = 0;
  (void)hipGetDevice(&device);
  if (device != device_) {
    const hipError_t error = hipSetDevice(device_); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevice:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevice\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorContextAlreadyInUse:
      (void)fprintf(stderr, "[%s@%s:%d] DeviceAlreadyInUse\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
  }
}

static void create_handle()
{
  status = hipsolverDnCreate(&handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ALLOC_FAILED:
    (void)fprintf(stderr, "[%s@%s:%d] ALLOC_FAILED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ARCH_MISMATCH:
    (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

static void alloc_gpu_mtx()
{
  size_t pitch = 0;
  const hipError_t err1 = hipMallocPitch(&Agpu, &pitch, Nmax * sizeof(cusolverDnX), Nmax); const int lin1 = __LINE__;
  switch (err1) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
    break;
  case hipErrorOutOfMemory:
    (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
    exit(err1);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, err1);
    exit(err1);
  }
  lda = int(pitch / sizeof(cusolverDnX));
#ifndef NDEBUG
  (void)fprintf(stdout, "lda = %d\n", lda);
#endif // !NDEBUG
  const hipError_t err2 = hipMemset2D(Agpu, pitch, 0, pitch, Nmax); const int lin2 = __LINE__;
  switch (err2) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
    break;
  case hipErrorInvalidValue:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, err2);
    exit(err2);
  }
}

static void find_lwork()
{
  int LworkL = -1, maxLworkL = 0;
  int LworkU = -1, maxLworkU = 0;

  for (int n = Nmin; n <= Nmax; n += Nstep) {
    status = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, Agpu, lda, &LworkL); const int lin1 = __LINE__;
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin1);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, status);
      exit(status);
    }
    if (LworkL > maxLworkL)
      maxLworkL = LworkL;
    status = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, Agpu, lda, &LworkU); const int lin2 = __LINE__;
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin2);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, status);
      exit(status);
    }
    if (LworkU > maxLworkU)
      maxLworkU = LworkU;
  }

  Lwork = ((maxLworkL >= maxLworkU) ? maxLworkL : maxLworkU);
#ifndef NDEBUG
  (void)fprintf(stdout, "Lwork = %d, LworkL = %d, LworkU = %d\n", Lwork, maxLworkL, maxLworkU);
#endif // !NDEBUG
}

static void alloc_gpu_wrk()
{
  if (Lwork > 0) {
    const hipError_t err1 = hipMalloc(&Workspace, Lwork * sizeof(cusolverDnX)); const int lin1 = __LINE__;
    switch (err1) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
      exit(err1);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, err1);
      exit(err1);
    }
    const hipError_t err2 = hipMemset(Workspace, 0, Lwork * sizeof(cusolverDnX)); const int lin2 = __LINE__;
    switch (err2) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
      exit(err2);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
      exit(err2);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, err2);
      exit(err2);
    }
  }
}

static void alloc_cpu_mtx()
{
  const size_t size = size_t(lda) * Nmax * sizeof(cusolverDnX);
  if (size > 0) {
    const hipError_t error = hipHostMalloc(&Acpu, size); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }
    (void)memset(Acpu, 0, size);
    wrk = (cusolverDnX*)calloc(3 * Nmax, sizeof(cusolverDnX)); const int lin2 = __LINE__;
    if (!wrk) {
      (void)fprintf(stderr, "[%s@%s:%d] ", __FUNCTION__, __FILE__, lin2);
      perror((const char*)NULL);
      exit(errno);
    }
  }
}

static double init_cpu_mtx()
{
  static const int idist = 1;
  int iseed[4] = { 0, 1, 2, 3 };
  const int k = Nmax - 1;
  int info = 0;

  const double go = omp_get_wtime();

  // diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, wrk);
  // A
  REAL_LAPACK(lagsy)(&Nmax, &k, wrk, Acpu, &lda, iseed, wrk + Nmax, &info); const int lin = __LINE__;
  if (info) {
    (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, info);
    exit(info);
  }

  return (omp_get_wtime() - go);
}

static double copy_mtx_cpu2gpu(const int n)
{
  const double go = omp_get_wtime();
  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const size_t pitch = lda * sizeof(cusolverDnX);
    const hipError_t error = hipMemcpy2D(Agpu, pitch, Acpu, pitch, n * sizeof(cusolverDnX), n, hipMemcpyHostToDevice); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidPitchValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidPitchValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidMemcpyDirection:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidMemcpyDirection\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }
    // just to be sure...
    (void)hipDeviceSynchronize();
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }
  return (omp_get_wtime() - go);
}

static double potrf_gpu(const bool upper, const int n)
{
  const double go = omp_get_wtime();
  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const hipblasFillMode_t uplo = (upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER);
    int devInfo = 0;
    status = hipsolverDnXpotrf(handle, uplo, n, Agpu, lda, Workspace, Lwork, &devInfo); const int lin = __LINE__;
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
      exit(status);
    }
    if (devInfo) {
      (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, devInfo);
      exit(devInfo);
    }
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }
  return (omp_get_wtime() - go);
}

static void free_cpu_mtx()
{
  if (Acpu) {
    const hipError_t error = hipHostFree(Acpu); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Acpu = (cusolverDnX*)NULL;
  }
}

static void free_gpu_wrk()
{
  if (Workspace) {
    const hipError_t error = hipFree(Workspace); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Workspace = (cusolverDnX*)NULL;
  }
}

static void free_gpu_mtx()
{
  const hipError_t error = hipFree(Agpu); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorNotInitialized:
    (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
    exit(error);    
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  Agpu = (cusolverDnX*)NULL;
}

static void destroy_handle()
{
  status = hipsolverDnDestroy(handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

int main(int argc, char* argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s Nmin Nmax Nstep #samples [device#]\n", *argv);
    return EXIT_FAILURE;
  }

  if ((Nmin = atoi(argv[1])) <= 0) {
    (void)fprintf(stderr, "Nmin == %d <= 0\n", Nmin);
    return EXIT_FAILURE;
  }

  if ((Nmax = atoi(argv[2])) <= 0) {
    (void)fprintf(stderr, "Nmax == %d <= 0\n", Nmax);
    return EXIT_FAILURE;
  }

  if (Nmax < Nmin) {
    (void)fprintf(stderr, "Nmax == %d < Nmin == %d\n", Nmax, Nmin);
    return EXIT_FAILURE;
  }

  if ((Nstep = atoi(argv[3])) <= 0) {
    (void)fprintf(stderr, "Nstep == %d <= 0\n", Nstep);
    return EXIT_FAILURE;
  }

  if ((_samples = atoi(argv[4])) <= 0) {
    (void)fprintf(stderr, "#samples == %d <= 0\n", _samples);
    return EXIT_FAILURE;
  }

  if ((argc == 6) && ((device_ = atoi(argv[5])) < 0)) {
    (void)fprintf(stderr, "device# == %d < 0\n", device_);
    return EXIT_FAILURE;
  }

#ifndef NDEBUG
  (void)fprintf(stdout, "[omp_get_wtick] %#.17E s\n", omp_get_wtick());
#endif // !NDEBUG

  device_count();
  if (device_ > _devices) {
    (void)fprintf(stderr, "device# == %d > #devices == %d\n", device_, _devices);
    return EXIT_FAILURE;
  }
  set_device();

  create_handle();
  alloc_gpu_mtx();
  find_lwork();
  alloc_gpu_wrk();
  alloc_cpu_mtx();
  const double init_time = init_cpu_mtx();
#ifndef NDEBUG
  (void)fprintf(stdout, "[init_cpu_mtx] %#.17E s\n", init_time);
#endif // !NDEBUG

  for (int n = Nmin; n <= Nmax; n += Nstep) {
    double Lcopy_times_min = INFINITY;
    double Lcopy_times_max = -0.0;
    double Lcopy_times_avg = -0.0;

    double Ucopy_times_min = INFINITY;
    double Ucopy_times_max = -0.0;
    double Ucopy_times_avg = -0.0;

    double Lpotrf_times_min = INFINITY;
    double Lpotrf_times_max = -0.0;
    double Lpotrf_times_avg = -0.0;

    double Upotrf_times_min = INFINITY;
    double Upotrf_times_max = -0.0;
    double Upotrf_times_avg = -0.0;

    for (int sample = 0; sample < _samples; ++sample) {
      const double Lcopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,L] %#.17E s\n", n, sample, Lcopy_time);
#endif // !NDEBUG
      if (Lcopy_time < Lcopy_times_min)
        Lcopy_times_min = Lcopy_time;
      if (Lcopy_time > Lcopy_times_max)
        Lcopy_times_max = Lcopy_time;
      Lcopy_times_avg += Lcopy_time / _samples;

      const double Lpotrf_time = potrf_gpu(false, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,L] %#.17E s\n", n, sample, Lpotrf_time);
#endif // !NDEBUG
      if (Lpotrf_time < Lpotrf_times_min)
        Lpotrf_times_min = Lpotrf_time;
      if (Lpotrf_time > Lpotrf_times_max)
        Lpotrf_times_max = Lpotrf_time;
      Lpotrf_times_avg += Lpotrf_time / _samples;

      const double Ucopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,U] %#.17E s\n", n, sample, Ucopy_time);
#endif // !NDEBUG
      if (Ucopy_time < Ucopy_times_min)
        Ucopy_times_min = Ucopy_time;
      if (Ucopy_time > Ucopy_times_max)
        Ucopy_times_max = Ucopy_time;
      Ucopy_times_avg += Ucopy_time / _samples;

      const double Upotrf_time = potrf_gpu(true, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,U] %#.17E s\n", n, sample, Upotrf_time);
#endif // !NDEBUG
      if (Upotrf_time < Upotrf_times_min)
        Upotrf_times_min = Upotrf_time;
      if (Upotrf_time > Upotrf_times_max)
        Upotrf_times_max = Upotrf_time;
      Upotrf_times_avg += Upotrf_time / _samples;
    }

    const double copy_times_min = ((Lcopy_times_min <= Ucopy_times_min) ? Lcopy_times_min : Ucopy_times_min);
    const double copy_times_max = ((Lcopy_times_max >= Ucopy_times_max) ? Lcopy_times_max : Ucopy_times_max);
    const double copy_times_avg = (Lcopy_times_avg + Ucopy_times_avg) / 2;

    (void)fprintf(stdout, "%d, %#.17E,%#.17E,%#.17E, %#.17E,%#.17E,%#.17E, %#.17E,%#.17E,%#.17E\n", n,
                  copy_times_min, copy_times_avg, copy_times_max,
                  Lpotrf_times_min, Lpotrf_times_avg, Lpotrf_times_max,
                  Upotrf_times_min, Upotrf_times_avg, Upotrf_times_max);
  }
  free_cpu_mtx();
  free_gpu_wrk();
  free_gpu_mtx();
  destroy_handle();

  return EXIT_SUCCESS;
}
