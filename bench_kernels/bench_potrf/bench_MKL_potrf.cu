#include "common.hpp"

// #include "mkl.h"

#ifdef USE_COMPLEX
#ifdef USE_FLOAT
#define dtype MKL_Complex8
#define btype float
#define Xpotrf CMPLX_LAPACK(potrf)
#else // USE_DOUBLE
#define dtype MKL_Complex16
#define btype double
#define Xpotrf CMPLX_LAPACK(potrf)
#endif // USE_FLOAT
#else // USE_REAL
#ifdef USE_FLOAT
#define dtype float
#define Xpotrf REAL_LAPACK(potrf)
#else // USE_DOUBLE
#define dtype double
#define Xpotrf REAL_LAPACK(potrf)
#endif // USE_FLOAT
#endif // USE_COMPLEX

static const char *const lin_fmt = "%d,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E\n";

static const int CACHE_LINE_BYTES = 64;
static const int CACHE_LINE_ELEMS = int(CACHE_LINE_BYTES / sizeof(dtype));

static int Nmin = 0, Nmax = 0, Nstep = 0, _samples = 0, lda = 0;
static dtype *A = (dtype*)NULL, *B = (dtype*)NULL;

//

static double alloc_cpu_mtx()
{
  const double go = omp_get_wtime();
  const int rem = Nmax % CACHE_LINE_ELEMS;
  lda = (rem ? (Nmax + (CACHE_LINE_ELEMS - rem)) : Nmax);
  const size_t siz1 = size_t(lda) * Nmax;
  const size_t size = 2 * siz1 * sizeof(dtype);
  errno = posix_memalign((void**)&A, CACHE_LINE_BYTES, size); const int lin = __LINE__;
  switch (errno) {
  case 0:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  default:
    (void)fprintf(stderr, "[%s@%s:%d,%d] ", __FUNCTION__, __FILE__, lin, errno);
    perror("posix_memalign");
    exit(errno);
  }
  B = A + siz1;
  const double end = (omp_get_wtime() - go);
  // don't time clearing the memory
  (void)memset(A, 0, size);
  return end;
}

static double init_cpu_mtx()
{
  const double go = omp_get_wtime();

  static const int idist = 1;
  int iseed[4] = { 0, 1, 2, 3 };
  const int k = Nmax - 1;
  int info = 0;

  dtype *const wrk = (dtype*)calloc(3 * Nmax, sizeof(dtype)); const int lin1 = __LINE__;
  if (!wrk) {
    (void)fprintf(stderr, "[%s@%s:%d,%d] ", __FUNCTION__, __FILE__, lin1, errno);
    perror("calloc");
    exit(errno);
  }

#ifdef USE_COMPLEX
  // Diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, (btype*)wrk);
  // A
  CMPLX_LAPACK(laghe)(&Nmax, &k, (btype*)wrk, A, &lda, iseed, wrk + Nmax, &info); const int lin2 = __LINE__;
#else // USE_REAL
  // Diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, wrk);
  // A
  REAL_LAPACK(lagsy)(&Nmax, &k, wrk, A, &lda, iseed, wrk + Nmax, &info); const int lin2 = __LINE__;
#endif // USE_COMPLEX
  if (info) {
    (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin2, info);
    exit(info);
  }

  free(wrk);
  return (omp_get_wtime() - go);
}

static double copy_mtx_cpu2cpu(const bool upper, const int n)
{
  const double go = omp_get_wtime();
#ifdef USE_COMPLEX
  CMPLX_LAPACK(lacpy)
#else // USE_REAL
  REAL_LAPACK(lacpy)
#endif // USE_COMPLEX
    (
#ifdef LACPY_ALL
     "A"
#else // U/L
     (upper ? "U" : "L")
#endif // LACPY_ALL
     , &n, &n, A, &lda, B, &lda);
  return (omp_get_wtime() - go);
}

static double potrf_cpu(const bool upper, const int n)
{
  const double go = omp_get_wtime();

  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    int info = 0;
    Xpotrf((upper ? "U" : "L"), &n, B, &lda, &info); const int lin = __LINE__;
    if (info) {
      (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, info);
      exit(info);
    }
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }

  return (omp_get_wtime() - go);
}

static double free_cpu_mtx()
{
  const double go = omp_get_wtime();

  if (A) {
    free(A);
    A = (dtype*)NULL;
  }

  return (omp_get_wtime() - go);
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    (void)fprintf(stderr, "%s Nmin Nmax Nstep #samples\n", *argv);
    return EXIT_FAILURE;
  }

  if ((Nmin = atoi(argv[1])) <= 0) {
    (void)fprintf(stderr, "Nmin == %d <= 0\n", Nmin);
    return EXIT_FAILURE;
  }

  if ((Nmax = atoi(argv[2])) <= 0) {
    (void)fprintf(stderr, "Nmax == %d <= 0\n", Nmax);
    return EXIT_FAILURE;
  }

  if (Nmax < Nmin) {
    (void)fprintf(stderr, "Nmax == %d < Nmin == %d\n", Nmax, Nmin);
    return EXIT_FAILURE;
  }

  if ((Nstep = atoi(argv[3])) <= 0) {
    (void)fprintf(stderr, "Nstep == %d <= 0\n", Nstep);
    return EXIT_FAILURE;
  }

  if ((_samples = atoi(argv[4])) <= 0) {
    (void)fprintf(stderr, "#samples == %d <= 0\n", _samples);
    return EXIT_FAILURE;
  }

  const char *const env_nthr = getenv("MKL_NUM_THREADS");
  if (!env_nthr) {
    (void)fprintf(stderr, "MKL_NUM_THREADS environment variable not set\n");
    return EXIT_FAILURE;
  }
  const int mkl_nthr = atoi(env_nthr);
  if (mkl_nthr <= 0) {
    (void)fprintf(stderr, "MKL_NUM_THREADS = %d <= 0\n", mkl_nthr);
    return EXIT_FAILURE;
  }

  const double resol = omp_get_wtick();
#ifndef NDEBUG
  (void)fprintf(stdout, "[omp_get_wtick] %#.17E s\n", resol);
#endif // !NDEBUG

  const double acpu_time = alloc_cpu_mtx();
  const double init_time = init_cpu_mtx();
#ifndef NDEBUG
  (void)fprintf(stdout, "[init_cpu_mtx] %#.17E s\n", init_time);
#endif // !NDEBUG

  (void)fprintf(stdout, "\"N\",\"COPY_H2H_MIN_s\",\"COPY_H2H_AVG_s\",\"COPY_H2H_MAX_s\",\"LPOTRF_MIN_s\",\"LPOTRF_AVG_s\",\"LPOTRF_MAX_s\",\"UPOTRF_MIN_s\",\"UPOTRF_AVG_s\",\"UPOTRF_MAX_s\"\n");
  (void)fflush(stdout);

  for (int n = Nmin; n <= Nmax; n += Nstep) {
    double Lcopy_times_min = INFINITY;
    double Lcopy_times_max = -0.0;
    double Lcopy_times_avg = -0.0;

    double Ucopy_times_min = INFINITY;
    double Ucopy_times_max = -0.0;
    double Ucopy_times_avg = -0.0;

    double Lpotrf_times_min = INFINITY;
    double Lpotrf_times_max = -0.0;
    double Lpotrf_times_avg = -0.0;

    double Upotrf_times_min = INFINITY;
    double Upotrf_times_max = -0.0;
    double Upotrf_times_avg = -0.0;

    for (int sample = 0; sample < _samples; ++sample) {
      const double Lcopy_time = copy_mtx_cpu2cpu(false, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2cpu(%d),%d,L] %#.17E s\n", n, sample, Lcopy_time);
#endif // !NDEBUG
      if (Lcopy_time < Lcopy_times_min)
        Lcopy_times_min = Lcopy_time;
      if (Lcopy_time > Lcopy_times_max)
        Lcopy_times_max = Lcopy_time;
      Lcopy_times_avg += Lcopy_time / _samples;

      const double Lpotrf_time = potrf_cpu(false, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_cpu(%d),%d,L] %#.17E s\n", n, sample, Lpotrf_time);
#endif // !NDEBUG
      if (Lpotrf_time < Lpotrf_times_min)
        Lpotrf_times_min = Lpotrf_time;
      if (Lpotrf_time > Lpotrf_times_max)
        Lpotrf_times_max = Lpotrf_time;
      Lpotrf_times_avg += Lpotrf_time / _samples;

      const double Ucopy_time = copy_mtx_cpu2cpu(true, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2cpu(%d),%d,U] %#.17E s\n", n, sample, Ucopy_time);
#endif // !NDEBUG
      if (Ucopy_time < Ucopy_times_min)
        Ucopy_times_min = Ucopy_time;
      if (Ucopy_time > Ucopy_times_max)
        Ucopy_times_max = Ucopy_time;
      Ucopy_times_avg += Ucopy_time / _samples;

      const double Upotrf_time = potrf_cpu(true, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_cpu(%d),%d,U] %#.17E s\n", n, sample, Upotrf_time);
#endif // !NDEBUG
      if (Upotrf_time < Upotrf_times_min)
        Upotrf_times_min = Upotrf_time;
      if (Upotrf_time > Upotrf_times_max)
        Upotrf_times_max = Upotrf_time;
      Upotrf_times_avg += Upotrf_time / _samples;
    }

    const double copy_times_min = ((Lcopy_times_min <= Ucopy_times_min) ? Lcopy_times_min : Ucopy_times_min);
    const double copy_times_max = ((Lcopy_times_max >= Ucopy_times_max) ? Lcopy_times_max : Ucopy_times_max);
    const double copy_times_avg = (Lcopy_times_avg + Ucopy_times_avg) / 2;

    (void)fprintf(stdout, lin_fmt, n,
                  copy_times_min, copy_times_avg, copy_times_max,
                  Lpotrf_times_min, Lpotrf_times_avg, Lpotrf_times_max,
                  Upotrf_times_min, Upotrf_times_avg, Upotrf_times_max);
    (void)fflush(stdout);
  }
  const double fcpu_time = free_cpu_mtx();
  const double agpu_time = -0.0;
  const double awrk_time = -0.0;
  const double fwrk_time = -0.0;
  const double fgpu_time = -0.0;

  (void)fprintf(stdout, lin_fmt, -_samples,
                resol, double(mkl_nthr), init_time,
                acpu_time, agpu_time, awrk_time,
                fcpu_time, fgpu_time, fwrk_time);
  (void)fflush(stdout);

  return EXIT_SUCCESS;
}
