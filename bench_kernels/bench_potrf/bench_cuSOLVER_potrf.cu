#include "common.hpp"

#include "hipsolver.h"

#ifdef USE_COMPLEX
#ifdef USE_FLOAT
#define dtype hipComplex
#define hipsolverDnXpotrf_bufferSize hipsolverDnCpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnCpotrf
#else // USE_DOUBLE
#define dtype hipDoubleComplex
#define hipsolverDnXpotrf_bufferSize hipsolverDnZpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnZpotrf
#endif // USE_FLOAT
#else // USE_REAL
#ifdef USE_FLOAT
#define dtype float
#define hipsolverDnXpotrf_bufferSize hipsolverDnSpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnSpotrf
#else // USE_DOUBLE
#define dtype double
#define hipsolverDnXpotrf_bufferSize hipsolverDnDpotrf_bufferSize
#define hipsolverDnXpotrf hipsolverDnDpotrf
#endif // USE_FLOAT
#endif // USE_COMPLEX

static int Nmin = 0, Nmax = 0, Nstep = 0, _samples = 0, device_ = 0, _devices = 0;
static int lda = 0, Lwork = 0;

static hipsolverHandle_t handle;

static dtype *Agpu = (dtype*)NULL, *Workspace = (dtype*)NULL, *Acpu = (dtype*)NULL, *wrk = (dtype*)NULL;

static void device_count()
{
  const hipError_t error = hipGetDeviceCount(&_devices); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorNoDevice:
    (void)fprintf(stderr, "[%s@%s:%d] NoDevice\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorInsufficientDriver:
    (void)fprintf(stderr, "[%s@%s:%d] InsufficientDriver\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
}

static void set_device()
{
  int device = 0;
  (void)hipGetDevice(&device);
  if (device != device_) {
    const hipError_t error = hipSetDevice(device_); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevice:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevice\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorContextAlreadyInUse:
      (void)fprintf(stderr, "[%s@%s:%d] DeviceAlreadyInUse\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
  }
}

static void create_handle()
{
  const hipsolverStatus_t status = hipsolverDnCreate(&handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ALLOC_FAILED:
    (void)fprintf(stderr, "[%s@%s:%d] ALLOC_FAILED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  case HIPSOLVER_STATUS_ARCH_MISMATCH:
    (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

static void alloc_gpu_mtx()
{
  size_t pitch = 0;
  const hipError_t err1 = hipMallocPitch(&Agpu, &pitch, Nmax * sizeof(dtype), Nmax); const int lin1 = __LINE__;
  switch (err1) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
    break;
  case hipErrorOutOfMemory:
    (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
    exit(err1);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, err1);
    exit(err1);
  }
  lda = int(pitch / sizeof(dtype));
#ifndef NDEBUG
  (void)fprintf(stdout, "lda = %d\n", lda);
#endif // !NDEBUG
  const hipError_t err2 = hipMemset2D(Agpu, pitch, 0, pitch, Nmax); const int lin2 = __LINE__;
  switch (err2) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
    break;
  case hipErrorInvalidValue:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, err2);
    exit(err2);
  }
}

static void find_lwork()
{
  int LworkL = -1, maxLworkL = 0;
  int LworkU = -1, maxLworkU = 0;

  for (int n = Nmin; n <= Nmax; n += Nstep) {
    const hipsolverStatus_t stat1 = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, Agpu, lda, &LworkL); const int lin1 = __LINE__;
    switch (stat1) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin1);
      exit(stat1);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin1);
      exit(stat1);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin1);
      exit(stat1);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin1);
      exit(stat1);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, stat1);
      exit(stat1);
    }
    if (LworkL > maxLworkL)
      maxLworkL = LworkL;
    const hipsolverStatus_t stat2 = hipsolverDnXpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, Agpu, lda, &LworkU); const int lin2 = __LINE__;
    switch (stat2) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin2);
      exit(stat2);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin2);
      exit(stat2);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin2);
      exit(stat2);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin2);
      exit(stat2);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, stat2);
      exit(stat2);
    }
    if (LworkU > maxLworkU)
      maxLworkU = LworkU;
  }

  Lwork = ((maxLworkL >= maxLworkU) ? maxLworkL : maxLworkU);
#ifndef NDEBUG
  (void)fprintf(stdout, "Lwork = %d, LworkL = %d, LworkU = %d\n", Lwork, maxLworkL, maxLworkU);
#endif // !NDEBUG
}

static void alloc_gpu_wrk()
{
  if (Lwork > 0) {
    const hipError_t err1 = hipMalloc(&Workspace, (Lwork + 1) * sizeof(dtype)); const int lin1 = __LINE__;
    switch (err1) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
      exit(err1);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, err1);
      exit(err1);
    }
    const hipError_t err2 = hipMemset(Workspace, 0, (Lwork + 1) * sizeof(dtype)); const int lin2 = __LINE__;
    switch (err2) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
      exit(err2);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
      exit(err2);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, err2);
      exit(err2);
    }
  }
}

static void alloc_cpu_mtx()
{
  const size_t size = size_t(lda) * Nmax * sizeof(dtype);
  if (size > 0) {
    const hipError_t error = hipHostMalloc(&Acpu, size); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorOutOfMemory:
      (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }
    (void)memset(Acpu, 0, size);
    wrk = (dtype*)calloc(3 * Nmax, sizeof(dtype)); const int lin2 = __LINE__;
    if (!wrk) {
      (void)fprintf(stderr, "[%s@%s:%d] ", __FUNCTION__, __FILE__, lin2);
      perror((const char*)NULL);
      exit(errno);
    }
  }
}

static double init_cpu_mtx()
{
  static const int idist = 1;
  int iseed[4] = { 0, 1, 2, 3 };
  const int k = Nmax - 1;
  int info = 0;

  const double go = omp_get_wtime();

  // diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, wrk);
  // A
  REAL_LAPACK(lagsy)(&Nmax, &k, wrk, Acpu, &lda, iseed, wrk + Nmax, &info); const int lin = __LINE__;
  if (info) {
    (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, info);
    exit(info);
  }

  return (omp_get_wtime() - go);
}

static double copy_mtx_cpu2gpu(const int n)
{
  const double go = omp_get_wtime();
  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const size_t pitch = lda * sizeof(dtype);
    const hipError_t error = hipMemcpy2D(Agpu, pitch, Acpu, pitch, n * sizeof(dtype), n, hipMemcpyHostToDevice); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidPitchValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidPitchValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidMemcpyDirection:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidMemcpyDirection\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }
    // just to be sure...
    (void)hipDeviceSynchronize();
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }
  return (omp_get_wtime() - go);
}

static double potrf_gpu(const bool upper, const int n)
{
  const double go = omp_get_wtime();
  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const hipblasFillMode_t uplo = (upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER);
    const hipsolverStatus_t status = hipsolverDnXpotrf(handle, uplo, n, Agpu, lda, Workspace, Lwork, (int*)(Workspace + Lwork)); const int lin1 = __LINE__;
    int devInfo = 0;
    (void)hipDeviceSynchronize();
    const hipError_t error = hipMemcpy(&devInfo, Workspace + Lwork, sizeof(int), hipMemcpyDeviceToHost); const int lin2 = __LINE__;
    (void)hipDeviceSynchronize();
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
      exit(error);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
      exit(error);
    case hipErrorInvalidMemcpyDirection:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidMemcpyDirection\n", __FUNCTION__, __FILE__, lin2);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, error);
      exit(error);      
    }
    switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d,%d] SUCCESS\n", __FUNCTION__, __FILE__, lin1, devInfo);
#endif // !NDEBUG
      break;
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      (void)fprintf(stderr, "[%s@%s:%d,%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin1, devInfo);
      exit(status);
    case HIPSOLVER_STATUS_INVALID_VALUE:
      (void)fprintf(stderr, "[%s@%s:%d,%d] INVALID_VALUE\n", __FUNCTION__, __FILE__, lin1, devInfo);
      exit(status);
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      (void)fprintf(stderr, "[%s@%s:%d,%d] ARCH_MISMATCH\n", __FUNCTION__, __FILE__, lin1, devInfo);
      exit(status);
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      (void)fprintf(stderr, "[%s@%s:%d,%d] INTERNAL_ERROR\n", __FUNCTION__, __FILE__, lin1, devInfo);
      exit(status);
    default:
      (void)fprintf(stderr, "[%s@%s:%d,%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, devInfo, status);
      exit(status);
    }
    if (devInfo) {
      (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin1, devInfo);
      exit(devInfo);
    }
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }
  return (omp_get_wtime() - go);
}

static void free_cpu_mtx()
{
  if (Acpu) {
    const hipError_t error = hipHostFree(Acpu); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Acpu = (dtype*)NULL;
  }
}

static void free_gpu_wrk()
{
  if (Workspace) {
    const hipError_t error = hipFree(Workspace); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Workspace = (dtype*)NULL;
  }
}

static void free_gpu_mtx()
{
  const hipError_t error = hipFree(Agpu); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorNotInitialized:
    (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
    exit(error);    
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  Agpu = (dtype*)NULL;
}

static void destroy_handle()
{
  const hipsolverStatus_t status = hipsolverDnDestroy(handle); const int lin = __LINE__;
  switch (status) {
  case HIPSOLVER_STATUS_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] SUCCESS\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case HIPSOLVER_STATUS_NOT_INITIALIZED:
    (void)fprintf(stderr, "[%s@%s:%d] NOT_INITIALIZED\n", __FUNCTION__, __FILE__, lin);
    exit(status);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, status);
    exit(status);
  }
}

int main(int argc, char* argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s Nmin Nmax Nstep #samples [device#]\n", *argv);
    return EXIT_FAILURE;
  }

  if ((Nmin = atoi(argv[1])) <= 0) {
    (void)fprintf(stderr, "Nmin == %d <= 0\n", Nmin);
    return EXIT_FAILURE;
  }

  if ((Nmax = atoi(argv[2])) <= 0) {
    (void)fprintf(stderr, "Nmax == %d <= 0\n", Nmax);
    return EXIT_FAILURE;
  }

  if (Nmax < Nmin) {
    (void)fprintf(stderr, "Nmax == %d < Nmin == %d\n", Nmax, Nmin);
    return EXIT_FAILURE;
  }

  if ((Nstep = atoi(argv[3])) <= 0) {
    (void)fprintf(stderr, "Nstep == %d <= 0\n", Nstep);
    return EXIT_FAILURE;
  }

  if ((_samples = atoi(argv[4])) <= 0) {
    (void)fprintf(stderr, "#samples == %d <= 0\n", _samples);
    return EXIT_FAILURE;
  }

  if ((argc == 6) && ((device_ = atoi(argv[5])) < 0)) {
    (void)fprintf(stderr, "device# == %d < 0\n", device_);
    return EXIT_FAILURE;
  }

#ifndef NDEBUG
  (void)fprintf(stdout, "[omp_get_wtick] %#.17E s\n", omp_get_wtick());
#endif // !NDEBUG

  device_count();
  if (device_ > _devices) {
    (void)fprintf(stderr, "device# == %d > #devices == %d\n", device_, _devices);
    return EXIT_FAILURE;
  }
  set_device();

  create_handle();
  alloc_gpu_mtx();
  find_lwork();
  alloc_gpu_wrk();
  alloc_cpu_mtx();
  const double init_time = init_cpu_mtx();
#ifndef NDEBUG
  (void)fprintf(stdout, "[init_cpu_mtx] %#.17E s\n", init_time);
#endif // !NDEBUG

  (void)fprintf(stdout, "\"N\",\"COPY_H2D_MIN_s\",\"COPY_H2D_AVG_s\",\"COPY_H2D_MAX_s\",\"LPOTRF_MIN_s\",\"LPOTRF_AVG_s\",\"LPOTRF_MAX_s\",\"UPOTRF_MIN_s\",\"UPOTRF_AVG_s\",\"UPOTRF_MAX_s\"\n");
  for (int n = Nmin; n <= Nmax; n += Nstep) {
    double Lcopy_times_min = INFINITY;
    double Lcopy_times_max = -0.0;
    double Lcopy_times_avg = -0.0;

    double Ucopy_times_min = INFINITY;
    double Ucopy_times_max = -0.0;
    double Ucopy_times_avg = -0.0;

    double Lpotrf_times_min = INFINITY;
    double Lpotrf_times_max = -0.0;
    double Lpotrf_times_avg = -0.0;

    double Upotrf_times_min = INFINITY;
    double Upotrf_times_max = -0.0;
    double Upotrf_times_avg = -0.0;

    for (int sample = 0; sample < _samples; ++sample) {
      const double Lcopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,L] %#.17E s\n", n, sample, Lcopy_time);
#endif // !NDEBUG
      if (Lcopy_time < Lcopy_times_min)
        Lcopy_times_min = Lcopy_time;
      if (Lcopy_time > Lcopy_times_max)
        Lcopy_times_max = Lcopy_time;
      Lcopy_times_avg += Lcopy_time / _samples;

      const double Lpotrf_time = potrf_gpu(false, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,L] %#.17E s\n", n, sample, Lpotrf_time);
#endif // !NDEBUG
      if (Lpotrf_time < Lpotrf_times_min)
        Lpotrf_times_min = Lpotrf_time;
      if (Lpotrf_time > Lpotrf_times_max)
        Lpotrf_times_max = Lpotrf_time;
      Lpotrf_times_avg += Lpotrf_time / _samples;

      const double Ucopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,U] %#.17E s\n", n, sample, Ucopy_time);
#endif // !NDEBUG
      if (Ucopy_time < Ucopy_times_min)
        Ucopy_times_min = Ucopy_time;
      if (Ucopy_time > Ucopy_times_max)
        Ucopy_times_max = Ucopy_time;
      Ucopy_times_avg += Ucopy_time / _samples;

      const double Upotrf_time = potrf_gpu(true, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,U] %#.17E s\n", n, sample, Upotrf_time);
#endif // !NDEBUG
      if (Upotrf_time < Upotrf_times_min)
        Upotrf_times_min = Upotrf_time;
      if (Upotrf_time > Upotrf_times_max)
        Upotrf_times_max = Upotrf_time;
      Upotrf_times_avg += Upotrf_time / _samples;
    }

    const double copy_times_min = ((Lcopy_times_min <= Ucopy_times_min) ? Lcopy_times_min : Ucopy_times_min);
    const double copy_times_max = ((Lcopy_times_max >= Ucopy_times_max) ? Lcopy_times_max : Ucopy_times_max);
    const double copy_times_avg = (Lcopy_times_avg + Ucopy_times_avg) / 2;

    (void)fprintf(stdout, "%d,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E\n", n,
                  copy_times_min, copy_times_avg, copy_times_max,
                  Lpotrf_times_min, Lpotrf_times_avg, Lpotrf_times_max,
                  Upotrf_times_min, Upotrf_times_avg, Upotrf_times_max);
  }
  free_cpu_mtx();
  free_gpu_wrk();
  free_gpu_mtx();
  destroy_handle();

  return EXIT_SUCCESS;
}
