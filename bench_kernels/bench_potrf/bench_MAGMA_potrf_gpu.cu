#include "common.hpp"

#include "magma.h"

#ifdef USE_COMPLEX
#ifdef USE_FLOAT
#define dtype magmaFloatComplex
#define btype float
#define magma_Xpotrf_gpu magma_cpotrf_gpu
#else // USE_DOUBLE
#define dtype magmaDoubleComplex
#define btype double
#define magma_Xpotrf_gpu magma_zpotrf_gpu
#endif // USE_FLOAT
#else // USE_REAL
#ifdef USE_FLOAT
#define dtype float
#define magma_Xpotrf_gpu magma_spotrf_gpu
#else // USE_DOUBLE
#define dtype double
#define magma_Xpotrf_gpu magma_dpotrf_gpu
#endif // USE_FLOAT
#endif // USE_COMPLEX

static const char *const lin_fmt = "%d,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E,%#.17E\n";

static int Nmin = 0, Nmax = 0, Nstep = 0, _samples = 0, device_ = 0, _devices = 0, lda = 0;
static dtype *Agpu = (dtype*)NULL, *Acpu = (dtype*)NULL;

//

static double device_count()
{
  const double go = omp_get_wtime();

  const hipError_t error = hipGetDeviceCount(&_devices); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorNoDevice:
    (void)fprintf(stderr, "[%s@%s:%d] NoDevice\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorInsufficientDriver:
    (void)fprintf(stderr, "[%s@%s:%d] InsufficientDriver\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }

  return (omp_get_wtime() - go);
}

static double set_device()
{
  const double go = omp_get_wtime();

  const magma_int_t err = magma_init(); const int lin = __LINE__;
  switch (err) {
  case MAGMA_SUCCESS:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case MAGMA_ERR_UNKNOWN:
    (void)fprintf(stderr, "[%s@%s:%d] ERR_UNKNOWN\n", __FUNCTION__, __FILE__, lin);
    return err;
  case MAGMA_ERR_HOST_ALLOC:
    (void)fprintf(stderr, "[%s@%s:%d] ERR_HOST_ALLOC\n", __FUNCTION__, __FILE__, lin);
    return err;
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, err);
    return err;
  }
  magma_setdevice(device_);

  return (omp_get_wtime() - go);
}

static double create_handle()
{
  const double go = omp_get_wtime();

  // nothing to create, but an opportunity to check how long does the timing itself take

  return (omp_get_wtime() - go);
}

static double alloc_gpu_mtx()
{
  const double go = omp_get_wtime();

  size_t pitch = 0;
  const hipError_t err1 = hipMallocPitch(&Agpu, &pitch, Nmax * sizeof(dtype), Nmax); const int lin1 = __LINE__;
  switch (err1) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
    break;
  case hipErrorOutOfMemory:
    (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin1);
    exit(err1);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, err1);
    exit(err1);
  }
  lda = int(pitch / sizeof(dtype));
#ifndef NDEBUG
  (void)fprintf(stdout, "lda = %d\n", lda);
#endif // !NDEBUG
  const double end = (omp_get_wtime() - go);
  // don't time clearing the memory
  const hipError_t err2 = hipMemset2D(Agpu, pitch, 0, pitch, Nmax); const int lin2 = __LINE__;
  switch (err2) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin2);
#endif // !NDEBUG
    break;
  case hipErrorInvalidValue:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin2);
    exit(err2);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin2, err2);
    exit(err2);
  }

  return end;
}

static double alloc_cpu_mtx()
{
  const double go = omp_get_wtime();

  const size_t size = size_t(lda) * Nmax * sizeof(dtype);
  const hipError_t error = hipHostMalloc(&Acpu, size); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorOutOfMemory:
    (void)fprintf(stderr, "[%s@%s:%d] MemoryAllocation\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  const double end = (omp_get_wtime() - go);
  // don't time clearing the memory
  (void)memset(Acpu, 0, size);

  return end;
}

static double init_cpu_mtx()
{
  const double go = omp_get_wtime();

  static const int idist = 1;
  int iseed[4] = { 0, 1, 2, 3 };
  const int k = Nmax - 1;
  int info = 0;

  dtype *const wrk = (dtype*)calloc(3 * Nmax, sizeof(dtype)); const int lin1 = __LINE__;
  if (!wrk) {
    (void)fprintf(stderr, "[%s@%s:%d,%d] ", __FUNCTION__, __FILE__, lin1, errno);
    perror("calloc");
    exit(errno);
  }

#ifdef USE_COMPLEX
  // Diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, (btype*)wrk);
  // Acpu
  CMPLX_LAPACK(laghe)(&Nmax, &k, (btype*)wrk, (MKL_Complex*)Acpu, &lda, iseed, (MKL_Complex*)(wrk + Nmax), &info); const int lin2 = __LINE__;
#else // USE_REAL
  // Diagonal
  REAL_LAPACK(larnv)(&idist, iseed, &Nmax, wrk);
  // Acpu
  REAL_LAPACK(lagsy)(&Nmax, &k, wrk, Acpu, &lda, iseed, wrk + Nmax, &info); const int lin2 = __LINE__;
#endif // USE_COMPLEX
  if (info) {
    (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin2, info);
    exit(info);
  }

  free(wrk);
  return (omp_get_wtime() - go);
}

static double copy_mtx_cpu2gpu(const int n)
{
  const double go = omp_get_wtime();

  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const size_t pitch = lda * sizeof(dtype);
    const hipError_t error = hipMemcpy2D(Agpu, pitch, Acpu, pitch, n * sizeof(dtype), n, hipMemcpyHostToDevice); const int lin1 = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin1);
#endif // !NDEBUG
      break;
    case hipErrorInvalidValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidPitchValue:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidPitchValue\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidDevicePointer:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    case hipErrorInvalidMemcpyDirection:
      (void)fprintf(stderr, "[%s@%s:%d] InvalidMemcpyDirection\n", __FUNCTION__, __FILE__, lin1);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin1, error);
      exit(error);
    }
    // just to be sure...
    (void)hipDeviceSynchronize();
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }

  return (omp_get_wtime() - go);
}

static double potrf_gpu(const bool upper, const int n)
{
  const double go = omp_get_wtime();

  if (n >= Nmin) {
    if (n > Nmax) {
      (void)fprintf(stderr, "[%s@%s] n == %d > Nmax == %d\n", __FUNCTION__, __FILE__, n, Nmax);
      exit(n);
    }
    const magma_uplo_t uplo = (upper ? MagmaUpper : MagmaLower);
    magma_int_t info = 0;
    const magma_int_t err = magma_Xpotrf_gpu(uplo, n, Agpu, lda, &info); const int lin = __LINE__;
    (void)hipDeviceSynchronize();
    switch (err) {
    case MAGMA_SUCCESS:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d,%d] SUCCESS\n", __FUNCTION__, __FILE__, lin, info);
#endif // !NDEBUG
      break;
    default:
      (void)fprintf(stderr, "[%s@%s:%d,%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, info, err);
      exit(err);
    }
    if (info) {
      (void)fprintf(stderr, "[%s@%s:%d] INFO = %d\n", __FUNCTION__, __FILE__, lin, info);
      exit(info);
    }
  }
  else {
    (void)fprintf(stderr, "[%s@%s] n == %d < Nmin == %d\n", __FUNCTION__, __FILE__, n, Nmin);
    exit(n);
  }

  return (omp_get_wtime() - go);
}

static double free_cpu_mtx()
{
  const double go = omp_get_wtime();

  if (Acpu) {
    const hipError_t error = hipHostFree(Acpu); const int lin = __LINE__;
    switch (error) {
    case hipSuccess:
#ifndef NDEBUG
      (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
      break;
    case hipErrorNotInitialized:
      (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
      exit(error);
    default:
      (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
      exit(error);
    }
    Acpu = (dtype*)NULL;
  }

  return (omp_get_wtime() - go);
}

static double free_gpu_mtx()
{
  const double go = omp_get_wtime();

  const hipError_t error = hipFree(Agpu); const int lin = __LINE__;
  switch (error) {
  case hipSuccess:
#ifndef NDEBUG
    (void)fprintf(stdout, "[%s@%s:%d] Success\n", __FUNCTION__, __FILE__, lin);
#endif // !NDEBUG
    break;
  case hipErrorInvalidDevicePointer:
    (void)fprintf(stderr, "[%s@%s:%d] InvalidDevicePointer\n", __FUNCTION__, __FILE__, lin);
    exit(error);
  case hipErrorNotInitialized:
    (void)fprintf(stderr, "[%s@%s:%d] InitializationError\n", __FUNCTION__, __FILE__, lin);
    exit(error);    
  default:
    (void)fprintf(stderr, "[%s@%s:%d] unknown error %d\n", __FUNCTION__, __FILE__, lin, error);
    exit(error);
  }
  Agpu = (dtype*)NULL;

  return (omp_get_wtime() - go);
}

static double destroy_handle()
{
  const double go = omp_get_wtime();

  magma_finalize();

  return (omp_get_wtime() - go);
}

int main(int argc, char* argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s Nmin Nmax Nstep #samples [device#]\n", *argv);
    return EXIT_FAILURE;
  }

  if ((Nmin = atoi(argv[1])) <= 0) {
    (void)fprintf(stderr, "Nmin == %d <= 0\n", Nmin);
    return EXIT_FAILURE;
  }

  if ((Nmax = atoi(argv[2])) <= 0) {
    (void)fprintf(stderr, "Nmax == %d <= 0\n", Nmax);
    return EXIT_FAILURE;
  }

  if (Nmax < Nmin) {
    (void)fprintf(stderr, "Nmax == %d < Nmin == %d\n", Nmax, Nmin);
    return EXIT_FAILURE;
  }

  if ((Nstep = atoi(argv[3])) <= 0) {
    (void)fprintf(stderr, "Nstep == %d <= 0\n", Nstep);
    return EXIT_FAILURE;
  }

  if ((_samples = atoi(argv[4])) <= 0) {
    (void)fprintf(stderr, "#samples == %d <= 0\n", _samples);
    return EXIT_FAILURE;
  }

  if ((argc == 6) && ((device_ = atoi(argv[5])) < 0)) {
    (void)fprintf(stderr, "device# == %d < 0\n", device_);
    return EXIT_FAILURE;
  }

  const char *const env_nthr = getenv("MKL_NUM_THREADS");
  if (!env_nthr) {
    (void)fprintf(stderr, "MKL_NUM_THREADS environment variable not set\n");
    return EXIT_FAILURE;
  }
  const int mkl_nthr = atoi(env_nthr);
  if (mkl_nthr <= 0) {
    (void)fprintf(stderr, "MKL_NUM_THREADS = %d <= 0\n", mkl_nthr);
    return EXIT_FAILURE;
  }

  const double resol = omp_get_wtick();
#ifndef NDEBUG
  (void)fprintf(stdout, "[omp_get_wtick] %#.17E s\n", resol);
#endif // !NDEBUG

  (void)device_count();
  if (device_ > _devices) {
    (void)fprintf(stderr, "device# == %d > #devices == %d\n", device_, _devices);
    return EXIT_FAILURE;
  }
  (void)set_device();

  (void)create_handle();
  const double agpu_time = alloc_gpu_mtx();
  const double acpu_time = alloc_cpu_mtx();
  const double init_time = init_cpu_mtx();
#ifndef NDEBUG
  (void)fprintf(stdout, "[init_cpu_mtx] %#.17E s\n", init_time);
#endif // !NDEBUG

  (void)fprintf(stdout, "\"N\",\"COPY_H2D_MIN_s\",\"COPY_H2D_AVG_s\",\"COPY_H2D_MAX_s\",\"LPOTRF_MIN_s\",\"LPOTRF_AVG_s\",\"LPOTRF_MAX_s\",\"UPOTRF_MIN_s\",\"UPOTRF_AVG_s\",\"UPOTRF_MAX_s\"\n");
  for (int n = Nmin; n <= Nmax; n += Nstep) {
    double Lcopy_times_min = INFINITY;
    double Lcopy_times_max = -0.0;
    double Lcopy_times_avg = -0.0;

    double Ucopy_times_min = INFINITY;
    double Ucopy_times_max = -0.0;
    double Ucopy_times_avg = -0.0;

    double Lpotrf_times_min = INFINITY;
    double Lpotrf_times_max = -0.0;
    double Lpotrf_times_avg = -0.0;

    double Upotrf_times_min = INFINITY;
    double Upotrf_times_max = -0.0;
    double Upotrf_times_avg = -0.0;

    for (int sample = 0; sample < _samples; ++sample) {
      const double Lcopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,L] %#.17E s\n", n, sample, Lcopy_time);
#endif // !NDEBUG
      if (Lcopy_time < Lcopy_times_min)
        Lcopy_times_min = Lcopy_time;
      if (Lcopy_time > Lcopy_times_max)
        Lcopy_times_max = Lcopy_time;
      Lcopy_times_avg += Lcopy_time / _samples;

      const double Lpotrf_time = potrf_gpu(false, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,L] %#.17E s\n", n, sample, Lpotrf_time);
#endif // !NDEBUG
      if (Lpotrf_time < Lpotrf_times_min)
        Lpotrf_times_min = Lpotrf_time;
      if (Lpotrf_time > Lpotrf_times_max)
        Lpotrf_times_max = Lpotrf_time;
      Lpotrf_times_avg += Lpotrf_time / _samples;

      const double Ucopy_time = copy_mtx_cpu2gpu(n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[copy_mtx_cpu2gpu(%d),%d,U] %#.17E s\n", n, sample, Ucopy_time);
#endif // !NDEBUG
      if (Ucopy_time < Ucopy_times_min)
        Ucopy_times_min = Ucopy_time;
      if (Ucopy_time > Ucopy_times_max)
        Ucopy_times_max = Ucopy_time;
      Ucopy_times_avg += Ucopy_time / _samples;

      const double Upotrf_time = potrf_gpu(true, n);
#ifndef NDEBUG
      (void)fprintf(stdout, "[potrf_gpu(%d),%d,U] %#.17E s\n", n, sample, Upotrf_time);
#endif // !NDEBUG
      if (Upotrf_time < Upotrf_times_min)
        Upotrf_times_min = Upotrf_time;
      if (Upotrf_time > Upotrf_times_max)
        Upotrf_times_max = Upotrf_time;
      Upotrf_times_avg += Upotrf_time / _samples;
    }

    const double copy_times_min = ((Lcopy_times_min <= Ucopy_times_min) ? Lcopy_times_min : Ucopy_times_min);
    const double copy_times_max = ((Lcopy_times_max >= Ucopy_times_max) ? Lcopy_times_max : Ucopy_times_max);
    const double copy_times_avg = (Lcopy_times_avg + Ucopy_times_avg) / 2;

    (void)fprintf(stdout, lin_fmt, n,
                  copy_times_min, copy_times_avg, copy_times_max,
                  Lpotrf_times_min, Lpotrf_times_avg, Lpotrf_times_max,
                  Upotrf_times_min, Upotrf_times_avg, Upotrf_times_max);
  }
  const double fcpu_time = free_cpu_mtx();
  const double fgpu_time = free_gpu_mtx();
  const double awrk_time = -0.0;
  const double fwrk_time = -0.0;
  (void)destroy_handle();

  (void)fprintf(stdout, lin_fmt, -_samples,
                resol, double(mkl_nthr), init_time,
                acpu_time, agpu_time, awrk_time,
                fcpu_time, fgpu_time, fwrk_time);

  return EXIT_SUCCESS;
}
